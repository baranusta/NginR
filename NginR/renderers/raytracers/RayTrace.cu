#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <iostream>

int elmSize;

struct vector3{
	float x;
	float y;
	float z;
	__device__ vector3(float X, float Y, float Z) : x(X), y(Y), z(Z)
	{}

	__device__ vector3()
	{}
	__device__ vector3 operator-(vector3 a)
	{
		vector3 result(x - a.x, y - a.y, z - a.z);
		return result;
	}
	__device__ vector3 operator+(vector3 a)
	{
		vector3 result(x + a.x, y + a.y, z + a.z);
		return result;
	}

	__device__ vector3 Normalize()
	{
		vector3 result;
		float length = sqrt(x*x + y*y + z*z);
		result.x = x / length;
		result.y = y / length;
		result.z = z / length;
		return result;
	}
	__device__ float dotProduct(vector3 t)
	{
		float result = x*t.x + y*t.y + z*t.z;
		return result;
	}
};

__device__ bool isIntersect(vector3 center, float r, vector3 ray, vector3 src, vector3& Normal, vector3 & Point, float &dist)
{
	float a = 1;
	float b = 2 * (ray.x *(src.x - center.x) + ray.y *(src.y - center.y) + ray.z *(src.z - center.z));
	float c = pow(src.x - center.x, 2) + pow(src.y - center.y, 2) + pow(src.z - center.z, 2) - r*r;
	float disc = b*b - 4 * c;
	if (disc<0)
	{
		return false;
	}
	disc = sqrt(disc);
	float t = (0.5f)*(-b - disc);
	if (t > 0)
	{
		dist = sqrt(a) * t;
		vector3 hitd;
		hitd.x = ray.x*t;
		hitd.y = ray.y*t;
		hitd.z = ray.z*t;
		Point.x = src.x + hitd.x;
		Point.y = src.y + hitd.y;
		Point.z = src.z + hitd.z;
		Normal.x = Point.x - center.x;
		Normal.y = Point.y - center.y;
		Normal.z = Point.z - center.z;
		return true;
	}
	else
		return false;
}
__device__
int rgbToInt2(float r, float g, float b)
{
	if (r > 1.0f)
		r = 1.0f;
	if (g > 1.0f)
		g = 1.0f;
	if (b > 1.0f)
		b = 1.0f;
	return (int(r * 255) << 16) | (int(g * 255) << 8) | int(b * 255);
}

__device__
int DetermineColor(float * spheres, vector3 ray, vector3 Normal, vector3 hitPoint, int objId, int size)
{
	float ka = 0.2f;
	vector3 Light;
	Light.x = 0;
	Light.y = 0;
	Light.z = 500;
	vector3 SourceToLight = Light - hitPoint;
	SourceToLight = SourceToLight.Normalize();
	Normal = Normal.Normalize();
	bool willBeShaded = false;

	for (int id = 0; id < size; id++)
	{
		vector3 N, hP;
		vector3 center;
		center.x = spheres[id * 8];
		center.y = spheres[id * 8 + 1];
		center.z = spheres[id * 8 + 2];
		float dist;
		if (id != objId && isIntersect(center, spheres[id * 8 + 3], SourceToLight, hitPoint, N, hP, dist))
		{
			willBeShaded = true;
			break;
		}
	}

	vector3 ambient;
	ambient.x = spheres[objId * 8 + 4];
	ambient.y = spheres[objId * 8 + 5];
	ambient.z = spheres[objId * 8 + 6];
	vector3 baseColor;
	baseColor.x = ambient.x*ka;
	baseColor.y = ambient.y*ka;
	baseColor.z = ambient.z*ka;
	if (willBeShaded)
	{
		return rgbToInt2(baseColor.x, baseColor.y, baseColor.z);
	}
	else{
		float val = SourceToLight.dotProduct(Normal);
		float kd = 0 < val ? val : 0;
		vector3 h = SourceToLight - ray;
		h = h.Normalize();
		val = h.dotProduct(Normal);
		float ks = pow(0< val ? val : 0, 20.0f);
		baseColor = baseColor + vector3(ambient.x*kd, ambient.y*kd, ambient.z*kd);
		baseColor = baseColor + vector3(ambient.x*ks, ambient.y*ks, ambient.z*ks);
		return rgbToInt2(baseColor.x, baseColor.y, baseColor.z);
	}
}

__global__
void cudaProcess(float * spheres, unsigned int *g_odata, int imgw, int imgh, int size)
{
	int tx = threadIdx.x;
	int bw = blockDim.x;
	int gx = gridDim.x;
	int x = blockIdx.x;
	for (int i = x; i < imgh; i += gx)
	{
		vector3 src;
		src.x = -5000;
		src.y = -(imgw / 2) + tx;
		src.z = -(imgh / 2) + i;

		vector3 dir;
		dir.x = 1;
		dir.y = 0;
		dir.z = 0;
		float minDist = 1000000;
		vector3 Normal, hitPoint;
		int objectId;
		for (int id = 0; id < size; id++)
		{
			float dist;
			vector3 N, hP;
			vector3 center;
			center.x = spheres[id * 8];
			center.y = spheres[id * 8 + 1];
			center.z = spheres[id * 8 + 2];

			float r = spheres[id * 8 + 3];
			if (isIntersect(center, r, dir, src, N, hP, dist))
			{
				if (dist < minDist)
				{
					minDist = dist;
					Normal = N;
					hitPoint = hP;
					objectId = id;
				}
			}
		}

		if (minDist != 1000000)
			g_odata[i*bw + tx] = DetermineColor(spheres, dir, Normal, hitPoint, objectId, size);
		else
			g_odata[i*bw + tx] = (int(0.4f * 255) << 16) | (int(0.4f * 255) << 8) | int(0.4f * 255);
	}
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
__global__
void cudaMoveObjects(float* spheres, int size)
{
	int id = threadIdx.x;
	if (id < size)
	{
		float dir = spheres[id * 8 + 7];
		if (spheres[id * 8 + 1] > 300 || spheres[id * 8 + 1] < -300)
		{
			dir *= -1;
			spheres[id * 8 + 7] = dir;
		}
		float velocity = 2.f * dir;
		if (id == 2)
		{
			spheres[id * 8 + 0] += velocity / 2;
			spheres[id * 8 + 2] += velocity / 2;
		}
		spheres[id * 8 + 1] += velocity;
	}
}

extern "C" void CudaUpdateObjects(float* SphereOnGpu,int elmSize)
{
	cudaMoveObjects << <1, 32 >> >(SphereOnGpu, elmSize);
}

extern "C" void CudaRayTrace(float* SphereOnGpu, unsigned int* out_data, int window_height2, int window_width2,int elmSize)
{
	cudaProcess << <256, window_width2 >> >(SphereOnGpu, out_data, window_width2, window_height2, elmSize);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
}


//extern "C" void CopyToGPU(SphereForGpu * SphereArr)
//{
//	hipMemcpy(SphereOnGpu, SphereArr, elmSize * sizeof(SphereForGpu), hipMemcpyHostToDevice);
//} 
//
//extern "C" void SetMemoryOnGPU(int size, SphereForGpu * SphereArr)
//{
//
//	elmSize = size;
//	hipMalloc((void**)&SphereOnGpu, size * sizeof(SphereForGpu));
//	CopyToGPU(SphereArr);
//}
//
//
//
//extern "C" void CopyToCpu(SphereForGpu * SphereArr)
//{
//	hipMemcpy(SphereArr, SphereOnGpu, elmSize * sizeof(SphereForGpu), hipMemcpyDeviceToHost);
//}
//
//extern "C" void FreeMemoryOnGPU()
//{
//	free(SphereOnGpu);
//}